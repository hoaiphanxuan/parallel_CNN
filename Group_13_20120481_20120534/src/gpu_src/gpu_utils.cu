#include "gpu_utils.h"

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}